#include <iostream>
#include <limits>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using std::cout;
using std::endl;

typedef unsigned long long Count;
typedef std::numeric_limits<double> DblLim;

const Count WARP_SIZE = 32; // Warp size
const Count NBLOCKS = 640; // Number of total cuda cores on my GPU
const Count ITERATIONS = 1000000; // Number of points to generate (each thread)

// This kernel is 
__global__ void picount(Count *totals) {
	// Define some shared memory: all threads in this block
	__shared__ Count counter[WARP_SIZE];

	// Unique ID of the thread
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// Initialize RNG
	hiprandState_t rng;
	hiprand_init(clock64(), tid, 0, &rng);

	// Initialize the counter
	counter[threadIdx.x] = 0;

	// Computation loop
	for (int i = 0; i < ITERATIONS; i++) {
		float x = hiprand_uniform(&rng); // Random x position in [0,1]
		float y = hiprand_uniform(&rng); // Random y position in [0,1]
		counter[threadIdx.x] += 1 - int(x * x + y * y); // Hit test
	}

	// The first thread in *every block* should sum the results
	if (threadIdx.x == 0) {
		// Reset count for this block
		totals[blockIdx.x] = 0;
		// Accumulate results
		for (int i = 0; i < WARP_SIZE; i++) {
			totals[blockIdx.x] += counter[i];
		}
	}
}

int main(int argc, char **argv) {
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		cout << "CUDA device missing! Do you need to use optirun?\n";
		return 1;
	}
	cout << "Starting simulation with " << NBLOCKS << " blocks, " << WARP_SIZE << " threads, and " << ITERATIONS << " iterations\n";

	// Allocate host and device memory to store the counters
	Count *hOut, *dOut;
	hOut = new Count[NBLOCKS]; // Host memory
	hipMalloc(&dOut, sizeof(Count) * NBLOCKS); // Device memory

	// Launch kernel
	picount<<<NBLOCKS, WARP_SIZE>>>(dOut);

	// Copy back memory used on device and free
	hipMemcpy(hOut, dOut, sizeof(Count) * NBLOCKS, hipMemcpyDeviceToHost);
	hipFree(dOut);

	// Compute total hits
	Count total = 0;
	for (int i = 0; i < NBLOCKS; i++) {
		total += hOut[i];
	}
	Count tests = NBLOCKS * ITERATIONS * WARP_SIZE;
	cout << "Approximated PI using " << tests << " random tests\n";

	// Set maximum precision for decimal printing
	cout.precision(DblLim::max_digits10);
	cout << "PI ~= " << 4.0 * (double)total/(double)tests << endl;

	return 0;
}
